#include "hip/hip_runtime.h"
//Zrównoleglenie - 1 blok - kilka wątków
#include<stdio.h>
#define N 32

__global__ void add(int *a, int *b, int *c) 
{
	//identyfikator wątku
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random (int *tab, int wym )
{	
	int i;
	for(i=0;i<wym;i++)
		tab[i]=rand()%101;
}


int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);
	int i;
	srand(time(NULL));
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random(a, N);
	b = (int *)malloc(size); random(b, N);
	c = (int *)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Uruchomienie kernela na urządzeniu CUDA - 1 blok - N wątków
	add<<<1,N>>>(d_a, d_b, d_c);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for(i=0;i<N;i++)
	{
		printf("a[%d](%d) + b[%d](%d) = c[%d](%d)\n",i,a[i],i,b[i],i,c[i]);
	}
	// Cleanup
	//printf("%d+%d=%d\n",a,b,c);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}


