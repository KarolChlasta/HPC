
#include <hip/hip_runtime.h>
#include<stdio.h>

//Funkcja uruchamiana na karcie graficznej (kernel)

__global__ void add(int *a, int *b, int *c) 
{
	*c = *a + *b;
}

int main(void) {
	int a, b, c; // Tablice przechowywane w pamieci hosta
	int *d_a, *d_b, *d_c; // Tablice przechowywane na urządzeniu CUDA
	int size = sizeof(int);
	// Alokacja pamieci na urządzeniu CUDA
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	// Przykłdowe wartości
	a = 2;
	b = 7;
	// Kopiowanie wartosci na urządzenie
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	// Uruchomienie kernela na urządzeniu CUDA - 1 blok - 1 wątek
	add<<<1,1>>>(d_a, d_b, d_c);
	// Kopiowanie uzyskanych wartości
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	// Czyszczenie
	printf("%d+%d=%d\n",a,b,c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
